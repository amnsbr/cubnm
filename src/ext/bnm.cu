#include "hip/hip_runtime.h"
/*
Reduced Wong-Wang model (Deco 2014) simulation on GPU

This code includes kernels needed for the simulation of BOLD signal and calculation of 
FC and FCD, in addition to other GPU-related functions.

Each simulation (for a given set of parameters) is run on a single GPU block, and
each thread in the block simulates one region of the brain. The threads in the block
form a "cooperative group" and are synchronized after each integration time step.
Calculation of FC and FCD are also similarly parallelized across simulations and region or
window pairs.

Parts of this code are based on https://github.com/BrainModes/The-Hybrid-Virtual-Brain, 
https://github.com/murraylab/hbnm & https://github.com/decolab/cb-neuromod

Author: Amin Saberi, Feb 2023
*/
#include "cubnm/includes.cuh"
#include "cubnm/defines.h"
#include "cubnm/utils.cuh"
#include "cubnm/models/bw.cuh"
#include "cubnm/models/base.cuh"
#include "cubnm/fc.cuh"
#include "cubnm/bnm.cuh"
#include "cubnm/models/rww.cuh"
#include "cubnm/models/rwwex.cuh"
#include "cubnm/models/kuramoto.cuh"
// other models go here

hipDeviceProp_t prop;

__device__ void global_input_cond(
        u_real& tmp_globalinput, int& k_buff_idx,
        const int& nodes, const int& sim_idx, const int& SC_idx,
        const int& j, int& k, int& buff_idx, u_real** SC, 
        int** delay, const bool& has_delay, const int& max_delay,
        u_real** conn_state_var_hist, u_real* conn_state_var_1
        ) {
    // calculates global input from other nodes `k` to current node `j`
    // Note: this will not skip over self-connections
    // if they should be ignored, their SC should be set to 0
    // Note 2: In SC and delay rows must be sources (k) and columns must
    // be targets (j), so that among threads of the same warp,
    // memory read is coalesced, such that at each k, SCs of all
    // k->j connections are adjacent in memory.
    // This is very important for performance especially
    // in higher number of nodes.
    tmp_globalinput = 0;
    if (has_delay) {
        for (k=0; k<nodes; k++) {
            // calculate correct index of the other region in the buffer based on j-k delay
            // buffer is moving backward, therefore the delay timesteps back in history
            // will be in +delay time steps in the buffer (then modulo max_delay as it is circular buffer)
            k_buff_idx = (buff_idx + delay[sim_idx][k*nodes+j]) % max_delay;
            tmp_globalinput += SC[SC_idx][k*nodes+j] * conn_state_var_hist[sim_idx][k_buff_idx*nodes+k];
        }
    } else {
        for (k=0; k<nodes; k++) {
            tmp_globalinput += SC[SC_idx][k*nodes+j] * conn_state_var_1[k];
        }            
    }
}

__device__ void global_input_osc(
        u_real& tmp_globalinput, int& k_buff_idx,
        const int& nodes, const int& sim_idx, const int& SC_idx,
        const int& j, int& k, int& buff_idx, u_real** SC, 
        int** delay, const bool& has_delay, const int& max_delay,
        u_real** conn_state_var_hist, u_real* conn_state_var_1
        ) {
    // calculates global input from other nodes `k` to current node `j`
    // See notes in global_input_cond
    tmp_globalinput = 0;
    if (has_delay) {
        for (k=0; k<nodes; k++) {
            // calculate correct index of the other region in the buffer based on j-k delay
            // buffer is moving backward, therefore the delay timesteps back in history
            // will be in +delay time steps in the buffer (then modulo max_delay as it is circular buffer)
            k_buff_idx = (buff_idx + delay[sim_idx][k*nodes+j]) % max_delay;
            tmp_globalinput += SC[SC_idx][k*nodes+j] * SIN(conn_state_var_hist[sim_idx][k_buff_idx*nodes+k] - conn_state_var_hist[sim_idx][buff_idx*nodes+j]);
        }
    } else {
        for (k=0; k<nodes; k++) {
            tmp_globalinput += SC[SC_idx][k*nodes+j] * SIN(conn_state_var_1[k] - conn_state_var_1[j]);
        }            
    }
}

template<typename Model>
__global__ void bnm(
        Model* model, u_real **BOLD, u_real ***states_out, 
        int **global_out_int, bool **global_out_bool,
        u_real **SC, int *SC_indices, 
        u_real **global_params, u_real **regional_params,
        u_real **conn_state_var_hist, int **delay, int max_delay,
        #ifdef NOISE_SEGMENT
        int *shuffled_nodes, int *shuffled_ts,
        #endif
        u_real *noise, uint* progress
    ) {
    // convert block to a cooperative group
    // get simulation and node indices
    int sim_idx = blockIdx.x;
    if (sim_idx >= model->N_SIMS) return;
    int j = threadIdx.x;
    if (j >= model->nodes) return;

    extern __shared__ u_real _shared_mem[];

    // copy variables used in the loop to local memory
    const int nodes = model->nodes;
    const int time_steps = model->time_steps;
    const int BOLD_TR = model->BOLD_TR;
    const int states_sampling = model->states_sampling;
    const bool ext_out = model->base_conf.ext_out;
    const bool states_ts = model->base_conf.states_ts;
    const bool sync_msec = model->base_conf.sync_msec;
    const int SC_idx = SC_indices[sim_idx];

    // set up noise shuffling if indicated
    #ifdef NOISE_SEGMENT
    /* 
    How noise shuffling works?
    At each time point we will have `ts_bold` which is the real time (in msec) 
    from the start of simulation, `ts_bold % noise_time_steps` which is the real 
    time passed within  each repeat of the noise segment (`curr_noise_repeat`), 
    `sh_ts_noise` which is the shuffled timepoint (column of the noise segment) 
    that will be used for getting the noise of nodes * 10 int_i * 2 neurons for 
    the current msec. 
    Similarly, in each thread we have `j` which is mapped to a `sh_j` which will 
    vary in each repeat.
    */
    const int noise_time_steps = model->base_conf.noise_time_steps;
    // get position of the node
    // in shuffled nodes for the first
    // repeat of noise segment
    int sh_j = shuffled_nodes[j];
    int curr_noise_repeat = 0;
    int sh_ts_noise = shuffled_ts[0];;
    #endif

    // determine the parameters of current simulation and node
    // use __shared__ for parameters that are shared
    // between regions in the same simulation, e.g. G, but
    // not for those that (may) vary, e.g. w_IE, w_EE and w_IE
    __shared__ u_real _global_params[Model::n_global_params];
    u_real _regional_params[Model::n_regional_params];
    int ii; // general-purpose index for parameters and varaiables
    for (ii=0; ii<Model::n_global_params; ii++) {
        _global_params[ii] = global_params[ii][sim_idx];
    }
    for (ii=0; ii<Model::n_regional_params; ii++) {
        _regional_params[ii] = regional_params[ii][sim_idx*nodes+j];
    }

    // initialize extended output sums
    if (ext_out && (!states_ts)) {
        for (ii=0; ii<Model::n_state_vars; ii++) {
            states_out[ii][sim_idx][j] = 0;
        }
    }

    // declare state variables, intermediate variables
    // and additional ints and bools
    u_real _state_vars[Model::n_state_vars];
    u_real _intermediate_vars[Model::n_intermediate_vars];
    // note: with this implementation n_state_vars and n_intermediate_vars
    // cannot be 0. Given they are frequently accessed it offers performance
    // improvement (as opposed to keeping them on heap). If a model does not
    // have any intermediate variables (having no states is undefined), 
    // n_intermediat_vars must be set to 1, but will be ignored in the model.
    // but keep the less frequently used variables on heap, while handling zero size
    int* _ext_int = (Model::n_ext_int > 0) ? (int*)malloc(Model::n_ext_int * sizeof(int)) : NULL;
    bool* _ext_bool = (Model::n_ext_bool > 0) ? (bool*)malloc(Model::n_ext_bool * sizeof(bool)) : NULL;
    int* _ext_int_shared = (int*)_shared_mem;
    bool* _ext_bool_shared = (bool*)(_shared_mem + Model::n_ext_int_shared*sizeof(int));
    // initialize model
    model->init(
        _state_vars, _intermediate_vars,
        _global_params, _regional_params,
        _ext_int, _ext_bool, 
        _ext_int_shared, _ext_bool_shared
    );


    // Ballon-Windkessel model variables
    u_real bw_x, bw_f, bw_nu, bw_q, tmp_f;
    bw_x = 0.0;
    bw_f = 1.0;
    bw_nu = 1.0;
    bw_q = 1.0;

    // delay setup
    const bool has_delay = (max_delay > 0);
    // if there is delay use a circular buffer (conn_state_var_hist)
    // and keep track of current buffer index (will be the same
    // in all nodes at each time point). Start from the end and
    // go backwards. 
    // Note that conn_state_var_hist is pseudo-2d
    int buff_idx, k_buff_idx;
    if (has_delay) {
        // initialize conn_state_var_hist in every time point at initial value
        for (buff_idx=0; buff_idx<max_delay; buff_idx++) {
            conn_state_var_hist[sim_idx][buff_idx*nodes+j] = _state_vars[Model::conn_state_var_idx];
        }
        buff_idx = max_delay-1;
    }

    // store immediate history of conn_state_var on extern shared memory
    // the memory is allocated dynamically based on the number of nodes
    // (see https://developer.nvidia.com/blog/using-shared-memory-cuda-cc/)
    u_real *_conn_state_var_1 = (u_real*)(_shared_mem + Model::n_ext_int_shared*sizeof(int)+Model::n_ext_bool_shared*sizeof(bool));
    u_real *conn_state_var_1;
    if (!(has_delay)) {
        // conn_state_var_1 is only used when
        // there is no delay
        if (nodes <= MAX_NODES_REG) {
            // for lower number of nodes
            // use shared memory for conn_state_var_1
            conn_state_var_1 = _conn_state_var_1;
        }
        #ifdef MANY_NODES
        else {
            // otherwise use global memory
            // allocated to conn_state_var_hist
            conn_state_var_1 = conn_state_var_hist[sim_idx];
        }
        // the else case only occurs if
        // MANY_NODES is defined but is also wrapped in
        // #ifdef MANY_NODES, as otherwise it'll hurt performance
        // of nodes <= MAX_NODES_REG simulations
        // (by making some compiler optimizations not possible)
        #endif
        conn_state_var_1[j] = _state_vars[Model::conn_state_var_idx];
    }

    // determine the global input function
    GlobalInputKernel global_input_kernel;
    if (Model::is_osc) {
        global_input_kernel = &global_input_osc;
    } else {
        global_input_kernel = &global_input_cond;
    }

    // this will determine if the simulation should be restarted
    // (e.g. if FIC adjustment fails in rWW)
    __shared__ bool restart;
    restart = false;

    // integration loop
    u_real tmp_globalinput = 0.0;
    int int_i = 0;
    int k = 0;
    long noise_idx = 0;
    int bold_i = 0;
    int states_i = 0;
    int ts_bold = 0;
    // outer loop for 1 msec steps
    // TODO: define number of steps for outer
    // and inner loops based on model dt and BW dt from user input 
    while (ts_bold < time_steps) {
        #ifdef NOISE_SEGMENT
        // get shuffled timepoint corresponding to
        // current noise repeat and the amount of time
        // past in the repeat
        sh_ts_noise = shuffled_ts[(ts_bold % noise_time_steps)+(curr_noise_repeat*noise_time_steps)];
        #endif
        if (sync_msec) {
            // calculate global input every 1 ms
            // (will be fixed through 10 steps of the millisecond)
            // note that a sync call is needed before using
            // and updating S_i_1_E, otherwise the current
            // thread might access S_E of other nodes at wrong
            // times (t or t-2 instead of t-1)
            __syncthreads();
            global_input_kernel(
                tmp_globalinput, k_buff_idx,
                nodes, sim_idx, SC_idx,
                j, k, buff_idx, SC, 
                delay, has_delay, max_delay,
                conn_state_var_hist, conn_state_var_1
            );
        }
        // inner loop for 0.1 msec steps
        for (int_i = 0; int_i < 10; int_i++) {
            if (!sync_msec) {
                // calculate global input every 0.1 ms
                __syncthreads();
                global_input_kernel(
                    tmp_globalinput, k_buff_idx,
                    nodes, sim_idx, SC_idx,
                    j, k, buff_idx, SC, 
                    delay, has_delay, max_delay,
                    conn_state_var_hist, conn_state_var_1
                );
            }
            // equations
            #ifdef NOISE_SEGMENT
            noise_idx = (((sh_ts_noise * 10 + int_i) * nodes * Model::n_noise) + (sh_j * Model::n_noise));
            #else
            noise_idx = (((ts_bold * 10 + int_i) * nodes * Model::n_noise) + (j * Model::n_noise));
            #endif
            model->step(
                _state_vars, _intermediate_vars, 
                _global_params, _regional_params,
                tmp_globalinput,
                noise, noise_idx
            );
            if (!sync_msec) {
                if (has_delay) {
                    // wait for other regions before updating so other
                    // nodes do not access S_i_E of current node at wrong times
                    __syncthreads();
                    conn_state_var_hist[sim_idx][buff_idx*nodes+j] = _state_vars[Model::conn_state_var_idx];
                    // go one time step backward in the buffer for the next time point
                    buff_idx = (buff_idx + max_delay - 1) % max_delay;
                    
                } else  {
                    // wait for other regions before updating so other
                    // nodes do not access S_i_E of current node at wrong times
                    __syncthreads();
                    conn_state_var_1[j] = _state_vars[Model::conn_state_var_idx];
                }
            }
        }

        if (sync_msec) {
            if (has_delay) {
                // wait for other regions before updating so other
                // nodes do not access S_i_E of current node at wrong times
                __syncthreads();
                conn_state_var_hist[sim_idx][buff_idx*nodes+j] = _state_vars[Model::conn_state_var_idx];
                // go one time step backward in the buffer for the next time point
                buff_idx = (buff_idx + max_delay - 1) % max_delay;
            } else {
                // wait for other regions before updating so other
                // nodes do not access S_i_E of current node at wrong times
                __syncthreads();
                conn_state_var_1[j] = _state_vars[Model::conn_state_var_idx];
            }
        }

        // Balloon-Windkessel model equations here since its
        // dt is 1 msec
        bw_step(bw_x, bw_f, bw_nu, 
            bw_q, tmp_f,
            _state_vars[Model::bold_state_var_idx]);

        // Calculate and write BOLD to managed memory every TR
        if ((ts_bold+1) % BOLD_TR == 0) {
            // calcualte and save BOLD
            BOLD[sim_idx][bold_i*nodes+j] = d_bwc.V_0_k1 * (1 - bw_q) + d_bwc.V_0_k2 * (1 - bw_q/bw_nu) + d_bwc.V_0_k3 * (1 - bw_nu);
            // update progress
            bold_i++;
            if (model->base_conf.verbose && (j==0)) {
                atomicAdd(progress, 1);
            }

        }

        if (ext_out) {
            if ((ts_bold+1) % states_sampling == 0) {
                // save time series of extended output if indicated
                if (states_ts) {
                    for (ii=0; ii<Model::n_state_vars; ii++) {
                        states_out[ii][sim_idx][states_i*nodes+j] = _state_vars[ii];
                    }
                } else {
                    // update sum (later mean) of extended
                    // output only after n_samples_remove_states
                    if (states_i >= model->n_states_samples_remove) {
                        for (ii=0; ii<Model::n_state_vars; ii++) {
                            states_out[ii][sim_idx][j] += _state_vars[ii];
                        }
                    }
                }
                states_i++;
            }
        }
        



        #ifdef NOISE_SEGMENT
        // reset noise segment time 
        // and shuffle nodes if the segment
        // has reached to the end
        if ((ts_bold+1) % noise_time_steps == 0) {
            // at the last time point don't do this
            // to avoid going over the extent of shuffled_nodes
            if (ts_bold+1 < time_steps) {
                curr_noise_repeat++;
                sh_j = shuffled_nodes[curr_noise_repeat*nodes+j];
            }
        }
        #endif

        if (Model::has_post_bw_step) {
            model->post_bw_step(
                _state_vars, _intermediate_vars,
                _ext_int, _ext_bool, 
                _ext_int_shared, _ext_bool_shared,
                restart,
                _global_params, _regional_params,
                ts_bold
            );
        }

        // move forward outer bw loop
        // this has to be before restart
        // because restart will reset ts_bold to 0
        ts_bold++;

        // if restart is indicated (e.g. FIC failed in rWW)
        // reset the simulation and start from the beginning
        if (restart) {
            // model-specific restart
            model->restart(
                _state_vars, _intermediate_vars, 
                _global_params, _regional_params,
                _ext_int, _ext_bool, 
                _ext_int_shared, _ext_bool_shared
            );
            // subtract progress of current simulation
            if (model->base_conf.verbose && (j==0)) {
                atomicAdd(progress, -bold_i);
            }
            // reset indices
            bold_i = 0;
            states_i = 0;
            ts_bold = 0;
            // reset Balloon-Windkessel model variables
            bw_x = 0.0;
            bw_f = 1.0;
            bw_nu = 1.0;
            bw_q = 1.0;
            if (has_delay) {
                // initialize conn_state_var_hist in every time point at initial value
                for (buff_idx=0; buff_idx<max_delay; buff_idx++) {
                    conn_state_var_hist[sim_idx][buff_idx*nodes+j] = _state_vars[Model::conn_state_var_idx];
                }
                // reset delay buffer index
                buff_idx = max_delay-1;
            } else {
                // reset conn_state_var_1
                conn_state_var_1[j] = _state_vars[Model::conn_state_var_idx];
            }
            #ifdef NOISE_SEGMENT
            // reset the node shuffling
            sh_j = shuffled_nodes[j];
            curr_noise_repeat = 0;
            #endif
            restart = false; // restart is done
            __syncthreads(); // make sure all threads are in sync after restart
        }
    }

    if (Model::has_post_integration) {
        model->post_integration(
            states_out, global_out_int, global_out_bool,
            _state_vars, _intermediate_vars, 
            _ext_int, _ext_bool, 
            _ext_int_shared, _ext_bool_shared,
            global_params, regional_params,
            _global_params, _regional_params,
            sim_idx, nodes, j
        );
    }
    if (ext_out && (!states_ts)) {
        // take average across time points after n_states_samples_remove
        int ext_out_time_points = states_i - model->n_states_samples_remove;
        for (ii=0; ii<Model::n_state_vars; ii++) {
            states_out[ii][sim_idx][j] /= ext_out_time_points;
        }
    }
    // free heap
    if (Model::n_ext_int > 0) {
        free(_ext_int);
    }
    if (Model::n_ext_bool > 0) {
        free(_ext_bool);
    }
}


template<typename Model>
__global__ void bnm_serial(
        Model* model, u_real **BOLD, u_real ***states_out, 
        int **global_out_int, bool **global_out_bool,
        u_real **SC, int *SC_indices, u_real **global_params, u_real **regional_params,
        u_real **conn_state_var_hist, int **delay, int max_delay,
        #ifdef NOISE_SEGMENT
        int *shuffled_nodes, int *shuffled_ts,
        #endif
        u_real *noise, uint* progress
    ) {
    // get simulation index
    int sim_idx = blockIdx.x;
    if (sim_idx >= model->N_SIMS) return;
    int j{0};

    // shared memory will be used for _ext_int and _ext_bool
    extern __shared__ u_real _shared_mem[];

    // copy variables used in the loop to local memory
    const int nodes = model->nodes;
    const int time_steps = model->time_steps;
    const int BOLD_TR = model->BOLD_TR;
    const bool ext_out = model->base_conf.ext_out;
    const bool states_ts = model->base_conf.states_ts;
    const bool sync_msec = model->base_conf.sync_msec;
    const int SC_idx = SC_indices[sim_idx];

    // set up noise shuffling if indicated
    #ifdef NOISE_SEGMENT
    const int noise_time_steps = model->base_conf.noise_time_steps;
    int sh_j{0};
    int curr_noise_repeat{0};
    int sh_ts_noise = shuffled_ts[0];
    #endif

    // copy parameters of current simulation to device heap memory
    u_real *_global_params = (u_real*)malloc(Model::n_global_params * sizeof(u_real));
    int ii; // general-purpose index for parameters and varaiables
    for (ii=0; ii<Model::n_global_params; ii++) {
        _global_params[ii] = global_params[ii][sim_idx];
    }
    u_real **_regional_params = (u_real**)malloc(nodes * sizeof(u_real*));
    for (j = 0; j < nodes; j++) {
        _regional_params[j] = (u_real*)malloc(Model::n_regional_params * sizeof(u_real));
        for (ii = 0; ii < Model::n_regional_params; ii++) {
            _regional_params[j][ii] = regional_params[ii][sim_idx * nodes + j];
        }
    }

    // initialize extended output sums
    if (ext_out && (!states_ts)) {
        for (ii=0; ii<Model::n_state_vars; ii++) {
            for (j=0; j<nodes; j++) {
                states_out[ii][sim_idx][j] = 0;
            }
        }
    }

    // allocated state variables, intermediate variables
    // and additional ints and bools on device heap
    u_real ** _state_vars = (u_real**)malloc(nodes * sizeof(u_real*));
    for (j=0; j<nodes; j++) {
        _state_vars[j] = (u_real*)malloc(Model::n_state_vars * sizeof(u_real));
    }
    u_real ** _intermediate_vars = (u_real**)malloc(nodes * sizeof(u_real*));
    for (j=0; j<nodes; j++) {
        _intermediate_vars[j] = (Model::n_intermediate_vars > 0) ? (u_real*)malloc(Model::n_intermediate_vars * sizeof(u_real)) : NULL;
    }
    int ** _ext_int = (int**)malloc(nodes * sizeof(int*));
    for (j=0; j<nodes; j++) {
        _ext_int[j] = (Model::n_ext_int > 0) ? (int*)malloc(Model::n_ext_int * sizeof(int)) : NULL;
    }
    bool ** _ext_bool = (bool**)malloc(nodes * sizeof(bool*));
    for (j=0; j<nodes; j++) {
        _ext_bool[j] = (Model::n_ext_bool > 0) ? (bool*)malloc(Model::n_ext_bool * sizeof(bool)) : NULL;
    }
    // shared variables    
    int* _ext_int_shared = (int*)_shared_mem;
    bool* _ext_bool_shared = (bool*)(_shared_mem + Model::n_ext_int_shared*sizeof(int));
    // initialize model
    for (j=0; j<nodes; j++) {
        model->init(
            _state_vars[j], _intermediate_vars[j],
            _global_params, _regional_params[j],
            _ext_int[j], _ext_bool[j], 
            _ext_int_shared, _ext_bool_shared
        );
    }


    // Ballon-Windkessel model variables
    u_real* bw_x = (u_real*)malloc(nodes * sizeof(u_real));
    u_real* bw_f = (u_real*)malloc(nodes * sizeof(u_real));
    u_real* bw_nu = (u_real*)malloc(nodes * sizeof(u_real));
    u_real* bw_q = (u_real*)malloc(nodes * sizeof(u_real));
    u_real tmp_f{0.0};
    for (j=0; j<nodes; j++) {
        bw_x[j] = 0.0;
        bw_f[j] = 1.0;
        bw_nu[j] = 1.0;
        bw_q[j] = 1.0;
    }

    // delay and conn_state history setup
    const bool has_delay = (max_delay > 0);
    // if there is delay use a circular buffer (conn_state_var_hist)
    // and keep track of current buffer index (will be the same
    // in all nodes at each time point). Start from the end and
    // go backwards. 
    // Note that conn_state_var_hist is pseudo-2d
    int buff_idx, k_buff_idx;
    u_real *conn_state_var_1; // immediate history when there is no delay
    if (has_delay) {
        for (j = 0; j < nodes; j++) {
            // initialize conn_state_var_hist in every time point at initial value
            for (buff_idx=0; buff_idx<max_delay; buff_idx++) {
                conn_state_var_hist[sim_idx][buff_idx*nodes+j] = _state_vars[j][Model::conn_state_var_idx];
            }
        }
        buff_idx = max_delay-1;
    } else {
        conn_state_var_1 = conn_state_var_hist[sim_idx];
        for (j = 0; j < nodes; j++) {
            conn_state_var_1[j] = _state_vars[j][Model::conn_state_var_idx];
        }
    }

    // determine the global input function
    GlobalInputKernel global_input_kernel;
    if (Model::is_osc) {
        global_input_kernel = &global_input_osc;
    } else {
        global_input_kernel = &global_input_cond;
    }
    
    // allocate memory for global input
    u_real * tmp_globalinput = (u_real*)malloc(nodes * sizeof(u_real));

    // this will determine if the simulation should be restarted
    // (e.g. if FIC adjustment fails in rWW)
    bool restart{false};

    // integration loop
    int int_i{0}, k{0}, bold_i{0}, ts_bold{0};
    long noise_idx{0};
    // outer loop for 1 msec steps
    // TODO: define number of steps for outer
    // and inner loops based on model dt and BW dt from user input 
    while (ts_bold < time_steps) {
        #ifdef NOISE_SEGMENT
        // get shuffled timepoint corresponding to
        // current noise repeat and the amount of time
        // past in the repeat
        sh_ts_noise = shuffled_ts[(ts_bold % noise_time_steps)+(curr_noise_repeat*noise_time_steps)];
        #endif
        if (sync_msec) {
            // calculate global input every 1 ms
            for (j=0; j<nodes; j++) {
                global_input_kernel(
                    tmp_globalinput[j], k_buff_idx,
                    nodes, sim_idx, SC_idx,
                    j, k, buff_idx, SC, 
                    delay, has_delay, max_delay,
                    conn_state_var_hist, conn_state_var_1
                );
            }
        }
        // inner loop for 0.1 msec steps
        for (int_i = 0; int_i < 10; int_i++) {
            if (!sync_msec) {
                // calculate global input every 0.1 ms
                for (j=0; j<nodes; j++) {
                    global_input_kernel(
                        tmp_globalinput[j], k_buff_idx,
                        nodes, sim_idx, SC_idx,
                        j, k, buff_idx, SC, 
                        delay, has_delay, max_delay,
                        conn_state_var_hist, conn_state_var_1
                    );
                }
            }
            // equations
            for (j=0; j<nodes; j++) {
                #ifdef NOISE_SEGMENT
                sh_j = shuffled_nodes[curr_noise_repeat*nodes+j];
                noise_idx = (((sh_ts_noise * 10 + int_i) * nodes * Model::n_noise) + (sh_j * Model::n_noise));
                #else
                noise_idx = (((ts_bold * 10 + int_i) * nodes * Model::n_noise) + (j * Model::n_noise));
                #endif
                model->step(
                    _state_vars[j], _intermediate_vars[j], 
                    _global_params, _regional_params[j],
                    tmp_globalinput[j],
                    noise, noise_idx
                );
            }
            if (!sync_msec) {
                if (has_delay) {
                    for (j=0; j<nodes; j++) {
                        conn_state_var_hist[sim_idx][buff_idx*nodes+j] = _state_vars[j][Model::conn_state_var_idx];
                    }
                    // go one time step backward in the buffer for the next time point
                    buff_idx = (buff_idx + max_delay - 1) % max_delay;
                    
                } else {
                    for (j=0; j<nodes; j++) {
                        conn_state_var_1[j] = _state_vars[j][Model::conn_state_var_idx];
                    }
                }
            }
        }

        if (sync_msec) {
            if (has_delay) {
                for (j=0; j<nodes; j++) {
                    conn_state_var_hist[sim_idx][buff_idx*nodes+j] = _state_vars[j][Model::conn_state_var_idx];
                }
                // go one time step backward in the buffer for the next time point
                buff_idx = (buff_idx + max_delay - 1) % max_delay;
                
            } else {
                for (j=0; j<nodes; j++) {
                    conn_state_var_1[j] = _state_vars[j][Model::conn_state_var_idx];
                }
            }
        }

        // Balloon-Windkessel model equations here since its
        // dt is 1 msec
        for (j=0; j<nodes; j++) {
            bw_step(bw_x[j], bw_f[j], bw_nu[j], 
                bw_q[j], tmp_f,
                _state_vars[j][Model::bold_state_var_idx]);
        }
        // Save BOLD and extended output to managed memory
        // every TR
        if ((ts_bold+1) % BOLD_TR == 0) {
            for (j=0; j<nodes; j++) {
                // calcualte and save BOLD
                BOLD[sim_idx][bold_i*nodes+j] = d_bwc.V_0_k1 * (1 - bw_q[j]) + d_bwc.V_0_k2 * (1 - bw_q[j]/bw_nu[j]) + d_bwc.V_0_k3 * (1 - bw_nu[j]);
                // save time series of extended output if indicated
                if (ext_out && states_ts) {
                    for (ii=0; ii<Model::n_state_vars; ii++) {
                        states_out[ii][sim_idx][bold_i*nodes+j] = _state_vars[j][ii];
                    }
                }
                // update sum (later mean) of extended
                // output only after n_vols_remove
                if ((bold_i>=model->n_vols_remove) && ext_out && (!states_ts)) {
                    for (ii=0; ii<Model::n_state_vars; ii++) {
                        states_out[ii][sim_idx][j] += _state_vars[j][ii];
                    }
                }
            }
            bold_i++;
            if (model->base_conf.verbose) {
                atomicAdd(progress, 1);
            }
        }

        #ifdef NOISE_SEGMENT
        // reset noise segment time 
        // and shuffle nodes if the segment
        // has reached to the end
        if ((ts_bold+1) % noise_time_steps == 0) {
            // at the last time point don't do this
            // to avoid going over the extent of shuffled_nodes
            if (ts_bold+1 < time_steps) {
                curr_noise_repeat++;
            }
        }
        #endif

        // Note: simulation restart
        // bool j_restart{false};
        // if (Model::has_post_bw_step) {
        //     for (j=0; j<nodes; j++) {
        //         j_restart = false;
        //         model->post_bw_step(
        //             _state_vars[j], _intermediate_vars[j],
        //             _ext_int[j], _ext_bool[j], 
        //             _ext_int_shared, _ext_bool_shared,
        //             j_restart,
        //             _global_params, _regional_params[j],
        //             ts_bold
        //         );
        //         restart = restart || j_restart;
        //     }
        // }

        // move forward outer bw loop
        // this has to be before restart
        // because restart will reset ts_bold to 0
        ts_bold++;

        // // if restart is indicated (e.g. FIC failed in rWW)
        // // reset the simulation and start from the beginning
        // if (restart) {
        //     // model-specific restart
        //     model->restart(_state_vars, _intermediate_vars, _ext_int, _ext_bool, _ext_int_shared, _ext_bool_shared);
        //     // subtract progress of current simulation
        //     if (model->base_conf.verbose && (j==0)) {
        //         atomicAdd(progress, -bold_i);
        //     }
        //     // reset indices
        //     bold_i = 0;
        //     ts_bold = 0;
        //     // reset Balloon-Windkessel model variables
        //     bw_x = 0.0;
        //     bw_f = 1.0;
        //     bw_nu = 1.0;
        //     bw_q = 1.0;
        //     if (has_delay) {
        //         // initialize conn_state_var_hist in every time point at initial value
        //         for (buff_idx=0; buff_idx<max_delay; buff_idx++) {
        //             conn_state_var_hist[sim_idx][buff_idx*nodes+j] = _state_vars[Model::conn_state_var_idx];
        //         }
        //         // reset delay buffer index
        //         buff_idx = max_delay-1;
        //     } else {
        //         // reset conn_state_var_1
        //         conn_state_var_1[j] = _state_vars[Model::conn_state_var_idx];
        //     }
        //     #ifdef NOISE_SEGMENT
        //     // reset the node shuffling
        //     sh_j = shuffled_nodes[j];
        //     curr_noise_repeat = 0;
        //     ts_noise = 0;
        //     sh_ts_noise = shuffled_ts[ts_noise];
        //     #endif
        //     restart = false; // restart is done
        //     __syncthreads(); // make sure all threads are in sync after restart
        // }
    }

    if (Model::has_post_integration) {
        for (j=0; j<nodes; j++) {
            model->post_integration(
                states_out, global_out_int, global_out_bool,
                _state_vars[j], _intermediate_vars[j], 
                _ext_int[j], _ext_bool[j], 
                _ext_int_shared, _ext_bool_shared,
                global_params, regional_params,
                _global_params, _regional_params[j],
                sim_idx, nodes, j
            );
        }
    }
    if (ext_out && (!states_ts)) {
        // take average across time points after n_vols_remove
        int ext_out_time_points = bold_i - model->n_vols_remove;
        for (ii=0; ii<Model::n_state_vars; ii++) {
            states_out[ii][sim_idx][j] /= ext_out_time_points;
        }
    }
    // free heap
    free(tmp_globalinput);
    free(bw_q);
    free(bw_nu);
    free(bw_f);
    free(bw_x);
    for (j=0; j<nodes; j++) {
        if (Model::n_ext_int > 0) {
            free(_ext_int[j]);
        }            
        if (Model::n_ext_bool > 0) {
            free(_ext_bool[j]);
        }
        free(_intermediate_vars[j]);
        free(_state_vars[j]);
        if (Model::n_regional_params > 0) {
            free(_regional_params[j]);
        }
    }
    free(_ext_int);
    free(_ext_bool);
    free(_intermediate_vars);
    free(_state_vars);
    free(_regional_params);
    if (Model::n_global_params > 0) {
        free(_global_params);
    }
}

template <typename Model>
void _run_simulations_gpu(
    double * BOLD_out, double * fc_trils_out, double * fcd_trils_out,
    u_real ** global_params, u_real ** regional_params, u_real * v_list,
    u_real **SC, int *SC_indices, u_real * SC_dist, BaseModel* m
)
{
    if (m->base_conf.verbose) {
        m->print_config();
    }

    // copy model to device 
    Model* h_model = (Model*)m; // cast BaseModel to its specific subclass, TODO: see if this is really needed
    Model* d_model;
    CUDA_CHECK_RETURN(hipMallocManaged(&d_model, sizeof(Model)));
    CUDA_CHECK_RETURN(hipMemcpy(d_model, h_model, sizeof(Model), hipMemcpyHostToDevice));

    // copy SC to managed memory
    for (int SC_idx=0; SC_idx<m->N_SCs; SC_idx++) {
      CUDA_CHECK_RETURN(hipMemcpy(m->d_SC[SC_idx], SC[SC_idx], m->nodes*m->nodes * sizeof(u_real), hipMemcpyHostToDevice));
    }
    // copy SC_indices to managed memory
    CUDA_CHECK_RETURN(hipMemcpy(m->d_SC_indices, SC_indices, m->N_SIMS * sizeof(int), hipMemcpyHostToDevice));

    // copy parameters to managed memory
    for (int i=0; i<Model::n_global_params; i++) {
        CUDA_CHECK_RETURN(hipMemcpy(m->d_global_params[i], global_params[i], m->N_SIMS * sizeof(u_real), hipMemcpyHostToDevice));
    }
    for (int i=0; i<Model::n_regional_params; i++) {
        CUDA_CHECK_RETURN(hipMemcpy(m->d_regional_params[i], regional_params[i], m->N_SIMS*m->nodes * sizeof(u_real), hipMemcpyHostToDevice));
    }

    // The following currently only does analytical FIC for rWW
    // but in theory can be used for any model that requires
    // parameter modifications
    // TODO: consider doing this in a separate function
    // called from Python, therefore final params are passed
    // to _run_simulations_gpu (except that they might be
    // modified during the simulation, e.g. in numerical FIC)
    m->prep_params(m->d_global_params, m->d_regional_params, v_list, 
        SC, SC_indices, SC_dist, 
        m->global_out_bool, m->global_out_int);

    // if indicated, calculate delay matrix of each simulation and allocate
    // memory to conn_state_var_hist according to the max_delay among the current simulations
    // Note: unlike many other variables delay and conn_state_var_hist are not global variables
    // and are not initialized in init_gpu, in order to allow variable ranges of velocities
    // in each run_simulations_gpu call within a session
    u_real **conn_state_var_hist; 
    CUDA_CHECK_RETURN(hipMallocManaged((void**)&conn_state_var_hist, sizeof(u_real*) * m->N_SIMS)); 
    int **delay;
    int max_delay{0}; // msec or 0.1 msec depending on base_conf.sync_msec; this is a global variable that will be used in the kernel
    float min_velocity{1e10}; // only used for printing info
    float max_length{0};
    float curr_length{0.0}, curr_velocity{0.0};
    int curr_delay{0};
    if (m->do_delay) {
    // note that do_delay is user asking for delay to be considered, has_delay indicates
    // if user has asked for delay AND there would be any delay between nodes given
    // velocity and distance matrix
    // TODO: make it less complicated
        CUDA_CHECK_RETURN(hipMallocManaged((void**)&delay, sizeof(int*) * m->N_SIMS));
        for (int sim_idx=0; sim_idx < m->N_SIMS; sim_idx++) {
            CUDA_CHECK_RETURN(hipMallocManaged((void**)&delay[sim_idx], sizeof(int) * m->nodes * m->nodes));
            curr_velocity = v_list[sim_idx];
            if (!m->base_conf.sync_msec) {
                curr_velocity /= 10;
            }
            if (curr_velocity < min_velocity) {
                min_velocity = curr_velocity;
            }
            for (int i = 0; i < m->nodes; i++) {
                for (int j = 0; j < m->nodes; j++) {
                    curr_length = SC_dist[i*m->nodes+j];
                    if (i > j) {
                        curr_delay = (int)round(curr_length/curr_velocity);
                        // set minimum delay to 1 because a node
                        // cannot access instantaneous states of 
                        // other nodes, as they might not have been
                        // calculated yet
                        curr_delay = std::max(curr_delay, 1);
                        delay[sim_idx][i*m->nodes + j] = curr_delay;
                        delay[sim_idx][j*m->nodes + i] = curr_delay;
                        if (curr_delay > max_delay) {
                            max_delay = curr_delay;
                            max_length = curr_length;
                        }
                    } else if (i == j) {
                        delay[sim_idx][i*m->nodes + j] = 1;
                    }
                }
            }
        }
    }
    bool has_delay = (max_delay > 0);
    if (has_delay) {
        if (m->base_conf.verbose) {
            std::string velocity_unit = "m/s";
            std::string delay_unit = "msec";
            if (!m->base_conf.sync_msec) {
                velocity_unit = "m/0.1s";
                delay_unit = "0.1msec";
            }
            std::cout << "Max distance " << max_length << " (mm) with a minimum velocity of " 
                << min_velocity << " (" << velocity_unit << ") => Max delay: " 
                << max_delay << " (" << delay_unit << ")" << std::endl;
        }
        // allocate memory to conn_state_var_hist for N_SIMS * (nodes * max_delay)
        // TODO: make it possible to have variable max_delay per each simulation
        for (int sim_idx=0; sim_idx < m->N_SIMS; sim_idx++) {
            CUDA_CHECK_RETURN(hipMallocManaged((void**)&conn_state_var_hist[sim_idx], sizeof(u_real) * m->nodes * max_delay));
        }
    }
    else if (
        (m->base_conf.serial)
        #ifdef MANY_NODES
        || (m->nodes > MAX_NODES_REG)
        #endif
    )    
     {
        // if there is no delay and the number of nodes is large
        // or nodes are simulated serially
        // store immediate history to conn_state_var_hist
        // on global memory, instead of shared memory
        for (int sim_idx=0; sim_idx < m->N_SIMS; sim_idx++) {
            CUDA_CHECK_RETURN(hipMallocManaged((void**)&conn_state_var_hist[sim_idx], sizeof(u_real) * m->nodes));
        }
    }

    // increase heap size as needed
    // start with initial heap size
    size_t heapSize = 0;
    CUDA_CHECK_RETURN(hipDeviceGetLimit(&heapSize, hipLimitMallocHeapSize));
    // add heap size required
    // in both serial and parallel nodes
    // _ext_int and _ext_bool are stored on heap
    heapSize += m->N_SIMS * (
        (Model::n_ext_int) * m->nodes * sizeof(int) +
        (Model::n_ext_bool) * m->nodes * sizeof(bool)
    );
    // in parallel case, these variables are also
    // on heap
    if (m->base_conf.serial) {
        heapSize += m->N_SIMS * (
            (Model::n_regional_params + Model::n_state_vars + Model::n_intermediate_vars) * m->nodes * sizeof(u_real) +
            (Model::n_global_params) * sizeof(u_real)
        );
    }
    CUDA_CHECK_RETURN(hipDeviceSetLimit(hipLimitMallocHeapSize, heapSize));

    // run simulations
    dim3 numBlocks(m->N_SIMS);
    dim3 threadsPerBlock(m->nodes);
    if (m->base_conf.serial) {
        threadsPerBlock.x = 1;
    }
    // calculate amount of required shared memory
    // used to store:
    // 1. _ext_int_shared
    // 2. _ext_bool_shared
    // 3. conn_state_var_1 if there is no delay
    // and the number of nodes is less than MAX_NODES_REG. When there is
    // delay this array is not needed. And with large number of
    // nodes there will be not enough shared memory available.
    size_t shared_mem_extern{0};
    shared_mem_extern += Model::n_ext_int_shared * sizeof(int) 
        + Model::n_ext_bool_shared * sizeof(bool);
    if ((!has_delay) && (m->nodes <= MAX_NODES_REG) && (!m->base_conf.serial)) {
        shared_mem_extern += m->nodes*sizeof(u_real);
    }
    // keep track of progress
    // Note: based on BOLD TRs reached in the first node
    // of each simulation (therefore the progress will be
    // an approximation of the real progress)
    uint* progress;
    CUDA_CHECK_RETURN(hipMallocManaged(&progress, sizeof(uint)));
    uint progress_final = m->bold_len * m->N_SIMS;
    *progress = 0;
    if (m->base_conf.serial) {
        bnm_serial<Model><<<numBlocks,threadsPerBlock,shared_mem_extern>>>(
            d_model,
            m->BOLD, m->states_out, 
            m->global_out_int,
            m->global_out_bool,
            m->d_SC, m->d_SC_indices,
            m->d_global_params, m->d_regional_params,
            conn_state_var_hist, delay, max_delay,
        #ifdef NOISE_SEGMENT
            m->shuffled_nodes, m->shuffled_ts,
        #endif
            m->noise, progress);
    } else {
        bnm<Model><<<numBlocks,threadsPerBlock,shared_mem_extern>>>(
            d_model,
            m->BOLD, m->states_out, 
            m->global_out_int,
            m->global_out_bool,
            m->d_SC, m->d_SC_indices,
            m->d_global_params, m->d_regional_params,
            conn_state_var_hist, delay, max_delay,
        #ifdef NOISE_SEGMENT
            m->shuffled_nodes, m->shuffled_ts,
        #endif
            m->noise, progress);
    }
    // asynchroneously print out the progress
    // if verbose
    if (m->base_conf.verbose) {
        uint last_progress = 0;
        uint no_progress_count = 0;
        while (*progress < progress_final) {
            // Print progress as percentage
            std::cout << std::fixed << std::setprecision(2) 
                << ((double)*progress / progress_final) * 100 << "%\r";
            std::cout.flush();
            // Sleep for interval ms
            std::this_thread::sleep_for(std::chrono::milliseconds(m->base_conf.progress_interval));
            // make sure it doesn't get stuck
            // by checking if there has been any progress
            if (*progress == last_progress) {
                no_progress_count++;
            } else {
                no_progress_count = 0;
            }
            if (no_progress_count > 50) {
                std::cout << "No progress detected in the last " << 50 * m->base_conf.progress_interval << " ms." << std::endl;
                break;
            }
            last_progress = *progress;
        }
        if (*progress == progress_final) {
            std::cout << "100.00%" << std::endl;
        } else {
            std::cout << "If no errors are shown, the simulation is still running "
                "but the progress is not being updated as there was no progress in the "
                "last " << m->base_conf.progress_interval <<  " ms, which may be too "
                "fast for current GPU and simulations" << std::endl;
        }
    }
    CUDA_CHECK_LAST_ERROR();
    CUDA_CHECK_RETURN(hipDeviceSynchronize());
    if (m->base_conf.verbose) {
        std::cout << "Simulation completed" << std::endl;
    }
    // calculate mean and sd bold for FC calculation
    threadsPerBlock.x = m->nodes;
    bold_stats<<<numBlocks, threadsPerBlock>>>(
        m->mean_bold, m->ssd_bold,
        m->BOLD, m->N_SIMS, m->nodes,
        m->bold_len, m->corr_len, m->n_vols_remove);
    CUDA_CHECK_LAST_ERROR();
    CUDA_CHECK_RETURN(hipDeviceSynchronize());
    // calculate window mean and sd bold for FCD calculations
    numBlocks.x = m->N_SIMS;
    numBlocks.y = m->n_windows;
    window_bold_stats<<<numBlocks,threadsPerBlock>>>(
        m->BOLD, m->N_SIMS, m->nodes, 
        m->n_windows, m->window_size+1, m->window_starts, m->window_ends,
        m->windows_mean_bold, m->windows_ssd_bold);
    CUDA_CHECK_LAST_ERROR();
    CUDA_CHECK_RETURN(hipDeviceSynchronize());
    // calculate FC and window FCs
    int maxThreadsPerBlock = prop.maxThreadsPerBlock;
    numBlocks.x = m->N_SIMS;
    numBlocks.y = ceil((float)m->n_pairs / (float)maxThreadsPerBlock);
    numBlocks.z = m->n_windows + 1; // +1 for total FC
    if (prop.maxThreadsPerBlock!=prop.maxThreadsDim[0]) {
        std::cerr << "Error: Code not implemented for GPUs in which maxThreadsPerBlock!=maxThreadsDim[0]" << std::endl;
        exit(1);
    }
    threadsPerBlock.x = maxThreadsPerBlock;
    threadsPerBlock.y = 1;
    threadsPerBlock.z = 1;
    fc<<<numBlocks, threadsPerBlock>>>(
        m->fc_trils, m->windows_fc_trils, m->BOLD, m->N_SIMS, m->nodes, m->n_pairs, 
        m->pairs_i, m->pairs_j,
        m->bold_len, m->n_vols_remove, m->corr_len, m->mean_bold, m->ssd_bold,
        m->n_windows, m->window_size+1, m->windows_mean_bold, m->windows_ssd_bold,
        m->window_starts, m->window_ends,
        maxThreadsPerBlock
    );
    CUDA_CHECK_LAST_ERROR();
    CUDA_CHECK_RETURN(hipDeviceSynchronize());
    // calculate window mean and sd fc_tril for FCD calculations
    numBlocks.x = m->N_SIMS;
    numBlocks.y = 1;
    numBlocks.z = 1;
    threadsPerBlock.x = m->n_windows;
    if (m->n_windows >= prop.maxThreadsPerBlock) {
        std::cerr << "Error: Mean/ssd FC tril of " << m->n_windows 
            << " windows cannot be calculated on this device" << std::endl;
        exit(1);
    }
    window_fc_stats<<<numBlocks,threadsPerBlock>>>(
        m->windows_mean_fc, m->windows_ssd_fc,
        NULL, NULL, NULL, NULL, // skipping L and R stats
        m->windows_fc_trils, m->N_SIMS, m->n_windows, m->n_pairs,
        false, 0);
    CUDA_CHECK_LAST_ERROR();
    CUDA_CHECK_RETURN(hipDeviceSynchronize());
    // calculate FCD
    numBlocks.x = m->N_SIMS;
    numBlocks.y = ceil((float)m->n_window_pairs / (float)maxThreadsPerBlock);
    numBlocks.z = 1;
    if (prop.maxThreadsPerBlock!=prop.maxThreadsDim[0]) {
        std::cerr << "Code not implemented for GPUs in which maxThreadsPerBlock!=maxThreadsDim[0]" << std::endl;
        exit(1);
    }
    threadsPerBlock.x = maxThreadsPerBlock;
    threadsPerBlock.y = 1;
    threadsPerBlock.z = 1;
    fcd<<<numBlocks, threadsPerBlock>>>(
        m->fcd_trils, NULL, NULL, // skipping separate L and R fcd
        m->windows_fc_trils, 
        m->windows_mean_fc, m->windows_ssd_fc,
        NULL, NULL, NULL, NULL,
        m->N_SIMS, m->n_pairs, m->n_windows, m->n_window_pairs, 
        m->window_pairs_i, m->window_pairs_j, maxThreadsPerBlock,
        false, 0);
    CUDA_CHECK_LAST_ERROR();
    CUDA_CHECK_RETURN(hipDeviceSynchronize());

    #ifdef USE_FLOATS
    // Convert FC and FCD to doubles for GOF calculation
    numBlocks.x = m->N_SIMS;
    numBlocks.y = m->n_pairs;
    numBlocks.z = 1;
    threadsPerBlock.x = 1;
    float2double<<<numBlocks, threadsPerBlock>>>(m->d_fc_trils, m->fc_trils, m->N_SIMS, m->n_pairs);
    CUDA_CHECK_LAST_ERROR();
    CUDA_CHECK_RETURN(hipDeviceSynchronize());
    numBlocks.x = m->N_SIMS;
    numBlocks.y = m->n_window_pairs;
    float2double<<<numBlocks, threadsPerBlock>>>(m->d_fcd_trils, m->fcd_trils, m->N_SIMS, m->n_window_pairs);
    CUDA_CHECK_LAST_ERROR();
    CUDA_CHECK_RETURN(hipDeviceSynchronize());
    #endif

    // copy the output from managed memory to _out arrays (which can be numpy arrays)
    size_t ext_out_size = m->nodes;
    if (m->base_conf.states_ts) {
        ext_out_size *= m->bold_len;
    }
    // TODO: pass the managed arrays data directly
    // to the python arrays without copying
    for (int sim_idx=0; sim_idx<m->N_SIMS; sim_idx++) {
        memcpy(BOLD_out, m->BOLD[sim_idx], sizeof(u_real) * m->bold_size);
        BOLD_out+=m->bold_size;
        memcpy(fc_trils_out, m->fc_trils[sim_idx], sizeof(u_real) * m->n_pairs);
        fc_trils_out+=m->n_pairs;
        memcpy(fcd_trils_out, m->fcd_trils[sim_idx], sizeof(u_real) * m->n_window_pairs);
        fcd_trils_out+=m->n_window_pairs;
    }
    if (m->modifies_params) { // e.g. rWW with FIC
        // copy (potentially) modified parameters back to the original array
        for (int i=0; i<Model::n_global_params; i++) {
            memcpy(global_params[i], m->d_global_params[i], m->N_SIMS * sizeof(u_real));
        }
        for (int i=0; i<Model::n_regional_params; i++) {
            memcpy(regional_params[i], m->d_regional_params[i], m->N_SIMS*m->nodes * sizeof(u_real));
        }
    }

    // free delay and conn_state_var_hist memories if allocated
    // Note: no need to clear memory of the other variables
    // as we'll want to reuse them in the next calls to run_simulations_gpu
    // within current session
    if (m->do_delay) {
        for (int sim_idx=0; sim_idx < m->N_SIMS; sim_idx++) {
            CUDA_CHECK_RETURN(hipFree(delay[sim_idx]));
        }
        CUDA_CHECK_RETURN(hipFree(delay));
    }
    if (has_delay) {
        for (int sim_idx=0; sim_idx < m->N_SIMS; sim_idx++) {
            CUDA_CHECK_RETURN(hipFree(conn_state_var_hist[sim_idx]));
        }
    }
    else if (
        (m->base_conf.serial)
        #ifdef MANY_NODES
        || (m->nodes > MAX_NODES_REG)
        #endif
    ) {
        for (int sim_idx=0; sim_idx < m->N_SIMS; sim_idx++) {
            CUDA_CHECK_RETURN(hipFree(conn_state_var_hist[sim_idx]));
        }
    } 
    CUDA_CHECK_RETURN(hipFree(conn_state_var_hist));
}

template <typename Model>
void _init_gpu(BaseModel *m, BWConstants bwc, bool force_reinit) {
    // check CUDA device avaliability and properties
    prop = get_device_prop(m->base_conf.verbose);

    // copy constants and configs from CPU
    // TODO: make these members of the model class
    CUDA_CHECK_RETURN(hipMemcpyToSymbol(HIP_SYMBOL(d_bwc), &bwc, sizeof(BWConstants)));
    if (strcmp(Model::name, "rWW")==0) {
        CUDA_CHECK_RETURN(hipMemcpyToSymbol(HIP_SYMBOL(d_rWWc), &Model::mc, sizeof(typename Model::Constants)));
    } 
    else if (strcmp(Model::name, "rWWEx")==0) {
        CUDA_CHECK_RETURN(hipMemcpyToSymbol(HIP_SYMBOL(d_rWWExc), &Model::mc, sizeof(typename Model::Constants)));
    }
    else if (strcmp(Model::name, "Kuramoto")==0) {
        CUDA_CHECK_RETURN(hipMemcpyToSymbol(HIP_SYMBOL(d_Kuramotoc), &Model::mc, sizeof(typename Model::Constants)));
    }

    // allocate device memory for SC
    CUDA_CHECK_RETURN(hipMallocManaged((void**)&(m->d_SC), sizeof(u_real*) * m->N_SCs));
    for (int SC_idx=0; SC_idx<m->N_SCs; SC_idx++) {
        CUDA_CHECK_RETURN(hipMallocManaged((void**)&(m->d_SC[SC_idx]), sizeof(u_real) * m->nodes*m->nodes));
    }
    CUDA_CHECK_RETURN(hipMallocManaged(&(m->d_SC_indices), sizeof(int) * m->N_SIMS));
 
    // allocate device memory for simulation parameters
    // size of global_params is (n_global_params, N_SIMS)
    // size of regional_params is (n_regional_params, N_SIMS * nodes)
    if (Model::n_global_params > 0) {
        CUDA_CHECK_RETURN(hipMallocManaged((void**)&(m->d_global_params), sizeof(u_real*) * Model::n_global_params));
        for (int param_idx=0; param_idx<Model::n_global_params; param_idx++) {
            CUDA_CHECK_RETURN(hipMallocManaged((void**)&(m->d_global_params[param_idx]), sizeof(u_real) * m->N_SIMS));
        }
    }
    if (Model::n_regional_params > 0) {
        CUDA_CHECK_RETURN(hipMallocManaged((void**)&(m->d_regional_params), sizeof(u_real*) * Model::n_regional_params));
        for (int param_idx=0; param_idx<Model::n_regional_params; param_idx++) {
            CUDA_CHECK_RETURN(hipMallocManaged((void**)&(m->d_regional_params[param_idx]), sizeof(u_real) * m->N_SIMS * m->nodes));
        }
    }

    // set up global int and bool outputs
    if (Model::n_global_out_int > 0) {
        CUDA_CHECK_RETURN(hipMallocManaged((void**)&(m->global_out_int), sizeof(int*) * Model::n_global_out_int));
        for (int i=0; i<Model::n_global_out_int; i++) {
            CUDA_CHECK_RETURN(hipMallocManaged((void**)&(m->global_out_int[i]), sizeof(int) * m->N_SIMS));
        }
    }
    if (Model::n_global_out_bool > 0) {
        CUDA_CHECK_RETURN(hipMallocManaged((void**)&(m->global_out_bool), sizeof(bool*) * Model::n_global_out_bool));
        for (int i=0; i<Model::n_global_out_bool; i++) {
            CUDA_CHECK_RETURN(hipMallocManaged((void**)&(m->global_out_bool[i]), sizeof(bool) * m->N_SIMS));
        }
    }

    // allocate memory for extended output
    size_t ext_out_size = m->nodes;
    if (m->base_conf.states_ts) {
        ext_out_size *= m->states_len;
    }
    if (m->base_conf.ext_out) {
        CUDA_CHECK_RETURN(hipMallocManaged((void**)&(m->states_out), sizeof(u_real**) * Model::n_state_vars));
        for (int var_idx=0; var_idx<Model::n_state_vars; var_idx++) {
            CUDA_CHECK_RETURN(hipMallocManaged((void**)&(m->states_out[var_idx]), sizeof(u_real*) * m->N_SIMS));
            for (int sim_idx=0; sim_idx<m->N_SIMS; sim_idx++) {
                CUDA_CHECK_RETURN(hipMallocManaged((void**)&(m->states_out[var_idx][sim_idx]), sizeof(u_real) * ext_out_size));
            }
        }
    }

    // specify n_vols_remove (for FC(D) calculations)
    m->n_vols_remove = m->base_conf.bold_remove_s * 1000 / m->BOLD_TR;

    // specifiy n_states_samples_remove (for states mean calculations)
    m->n_states_samples_remove = m->base_conf.bold_remove_s * 1000 / m->states_sampling;

    // preparing FC calculations
    m->corr_len = m->bold_len - m->n_vols_remove;
    if (m->corr_len < 2) {
        std::cerr << "Number of BOLD volumes (after removing initial volumes) is too low for FC calculations" << std::endl;
        exit(1);
    }
    CUDA_CHECK_RETURN(hipMallocManaged((void**)&(m->BOLD), sizeof(u_real*) * m->N_SIMS));
    CUDA_CHECK_RETURN(hipMallocManaged((void**)&(m->mean_bold), sizeof(u_real*) * m->N_SIMS));
    CUDA_CHECK_RETURN(hipMallocManaged((void**)&(m->ssd_bold), sizeof(u_real*) * m->N_SIMS));
    m->n_pairs = ((m->nodes) * (m->nodes - 1)) / 2;
    int rh_idx;
    if (m->base_conf.exc_interhemispheric) {
        if ((m->nodes % 2) != 0) {
            std::cerr << "Error: exc_interhemispheric is set but number of nodes is not even" << std::endl;
            exit(1);
        }
        rh_idx = m->nodes / 2; // assumes symmetric number of parcels and L->R order
        m->n_pairs -= pow(rh_idx, 2); // exclude the middle square
    }
    // create a mapping between pair_idx and i and j
    int curr_idx = 0;
    CUDA_CHECK_RETURN(hipMallocManaged(&(m->pairs_i), sizeof(int) * m->n_pairs));
    CUDA_CHECK_RETURN(hipMallocManaged(&(m->pairs_j), sizeof(int) * m->n_pairs));
    for (int i=0; i < m->nodes; i++) {
        for (int j=0; j < m->nodes; j++) {
            if (i > j) {
                if (m->base_conf.exc_interhemispheric) {
                    // skip if each node belongs to a different hemisphere
                    if ((i < rh_idx) ^ (j < rh_idx)) {
                        continue;
                    }
                }
                m->pairs_i[curr_idx] = i;
                m->pairs_j[curr_idx] = j;
                curr_idx++;
            }
        }
    }
    // allocate memory for fc trils
    CUDA_CHECK_RETURN(hipMallocManaged((void**)&(m->fc_trils), sizeof(u_real*) * m->N_SIMS));

    // FCD preparation
    // calculate number of windows and window start/end indices
    int *_window_starts, *_window_ends; // are cpu integer arrays
    m->n_windows = get_dfc_windows(
        &_window_starts, &_window_ends, 
        m->corr_len, m->bold_len, m->n_vols_remove,
        m->window_step, m->window_size, m->base_conf.drop_edges);
    if (m->n_windows == 0) {
        std::cerr << "Error: Number of windows is 0" << std::endl;
        exit(1);
    }
    CUDA_CHECK_RETURN(hipMallocManaged(&(m->window_starts), sizeof(int) * m->n_windows));
    CUDA_CHECK_RETURN(hipMallocManaged(&(m->window_ends), sizeof(int) * m->n_windows));
    for (int i=0; i<m->n_windows; i++) {
        m->window_starts[i] = _window_starts[i];
        m->window_ends[i] = _window_ends[i];
    }
    // allocate memory for mean and ssd BOLD of each window
    // (n_sims x n_windows x nodes)
    CUDA_CHECK_RETURN(hipMallocManaged((void**)&(m->windows_mean_bold), sizeof(u_real*) * m->N_SIMS));
    CUDA_CHECK_RETURN(hipMallocManaged((void**)&(m->windows_ssd_bold), sizeof(u_real*) * m->N_SIMS));
    CUDA_CHECK_RETURN(hipMallocManaged((void**)&(m->windows_fc_trils), sizeof(u_real*) * m->N_SIMS));
    // allocate memory for mean and ssd fc_tril of each window
    // (n_sims x n_windows)
    CUDA_CHECK_RETURN(hipMallocManaged((void**)&(m->windows_mean_fc), sizeof(u_real*) * m->N_SIMS));
    CUDA_CHECK_RETURN(hipMallocManaged((void**)&(m->windows_ssd_fc), sizeof(u_real*) * m->N_SIMS));
    // create a mapping between window_pair_idx and i and j
    m->n_window_pairs = (m->n_windows * (m->n_windows-1)) / 2;
    curr_idx = 0;
    CUDA_CHECK_RETURN(hipMallocManaged(&(m->window_pairs_i), sizeof(int) * m->n_window_pairs));
    CUDA_CHECK_RETURN(hipMallocManaged(&(m->window_pairs_j), sizeof(int) * m->n_window_pairs));
    for (int i=0; i < m->n_windows; i++) {
        for (int j=0; j < m->n_windows; j++) {
            if (i > j) {
                m->window_pairs_i[curr_idx] = i;
                m->window_pairs_j[curr_idx] = j;
                curr_idx++;
            }
        }
    }
    // allocate memory for fcd trils
    CUDA_CHECK_RETURN(hipMallocManaged((void**)&(m->fcd_trils), sizeof(u_real*) * m->N_SIMS));

    #ifdef USE_FLOATS
    // allocate memory for double versions of fc and fcd trils on CPU
    CUDA_CHECK_RETURN(hipMallocManaged((void**)&(m->d_fc_trils), sizeof(double*) * m->N_SIMS));
    CUDA_CHECK_RETURN(hipMallocManaged((void**)&(m->d_fcd_trils), sizeof(double*) * m->N_SIMS));
    #else
    // use d_fc_trils and d_fcd_trils as aliases for fc_trils and fcd_trils
    m->d_fc_trils = m->fc_trils;
    m->d_fcd_trils = m->fcd_trils;
    #endif



    // allocate memory per each simulation
    for (int sim_idx=0; sim_idx<m->N_SIMS; sim_idx++) {
        // allocate a chunk of BOLD to this simulation (not sure entirely if this is the best way to do it)
        CUDA_CHECK_RETURN(hipMallocManaged((void**)&(m->BOLD[sim_idx]), sizeof(u_real) * m->bold_size));
        // allocate memory for fc calculations
        CUDA_CHECK_RETURN(hipMallocManaged((void**)&(m->mean_bold[sim_idx]), sizeof(u_real) * m->nodes));
        CUDA_CHECK_RETURN(hipMallocManaged((void**)&(m->ssd_bold[sim_idx]), sizeof(u_real) * m->nodes));
        CUDA_CHECK_RETURN(hipMallocManaged((void**)&(m->fc_trils[sim_idx]), sizeof(u_real) * m->n_pairs));
        // allocate memory for window fc and fcd calculations
        CUDA_CHECK_RETURN(hipMallocManaged((void**)&(m->windows_mean_bold[sim_idx]), sizeof(u_real) * m->n_windows * m->nodes));
        CUDA_CHECK_RETURN(hipMallocManaged((void**)&(m->windows_ssd_bold[sim_idx]), sizeof(u_real) * m->n_windows * m->nodes));
        CUDA_CHECK_RETURN(hipMallocManaged((void**)&(m->windows_fc_trils[sim_idx]), sizeof(u_real) * m->n_windows * m->n_pairs));
        CUDA_CHECK_RETURN(hipMallocManaged((void**)&(m->windows_mean_fc[sim_idx]), sizeof(u_real) * m->n_windows));
        CUDA_CHECK_RETURN(hipMallocManaged((void**)&(m->windows_ssd_fc[sim_idx]), sizeof(u_real) * m->n_windows));
        CUDA_CHECK_RETURN(hipMallocManaged((void**)&(m->fcd_trils[sim_idx]), sizeof(u_real) * m->n_window_pairs));
        #ifdef USE_FLOATS
        // allocate memory for double copies of fc and fcd
        CUDA_CHECK_RETURN(hipMallocManaged((void**)&(m->d_fc_trils[sim_idx]), sizeof(double) * m->n_pairs));
        CUDA_CHECK_RETURN(hipMallocManaged((void**)&(m->d_fcd_trils[sim_idx]), sizeof(double) * m->n_window_pairs));
        #endif
    }

    // check if noise needs to be calculated
    if (
        (m->rand_seed != m->last_rand_seed) ||
        (m->time_steps != m->last_time_steps) ||
        (m->nodes != m->last_nodes) ||
        (m->base_conf.noise_time_steps != m->last_noise_time_steps) ||
        (!m->gpu_initialized) ||
        force_reinit
        ) {
        // pre-calculate normally-distributed noise on CPU
        // this is necessary to ensure consistency of noise given the same seed
        // doing the same thing directly on the device is more challenging
        #ifndef NOISE_SEGMENT
        // precalculate the entire noise needed; can use up a lot of memory
        // with high N of nodes and longer durations leads maxes out the memory
        m->noise_size = m->nodes * m->time_steps * 10 * Model::n_noise; // *10 for 0.1msec
        #else
        // otherwise precalculate a noise segment and arrays of shuffled
        // nodes and time points and reuse-shuffle the noise segment
        // throughout the simulation for `noise_repeats`
        m->noise_size = m->nodes * (m->base_conf.noise_time_steps) * 10 * Model::n_noise;
        m->noise_repeats = ceil((float)(m->time_steps) / (float)(m->base_conf.noise_time_steps));
        #endif
        if (m->base_conf.verbose) {
            std::cout << "Precalculating " << m->noise_size << " noise elements..." << std::endl;
        }
        if (m->last_nodes != 0) {
            // noise is being recalculated, free the previous one
            CUDA_CHECK_RETURN(hipFree(m->noise));
            #ifdef NOISE_SEGMENT
            CUDA_CHECK_RETURN(hipFree(m->shuffled_nodes));
            CUDA_CHECK_RETURN(hipFree(m->shuffled_ts));
            #endif
        }
        m->last_time_steps = m->time_steps;
        m->last_nodes = m->nodes;
        m->last_rand_seed = m->rand_seed;
        m->last_noise_time_steps = m->base_conf.noise_time_steps;
        std::mt19937 rand_gen(m->rand_seed);
        std::normal_distribution<float> normal_dist(0, 1);
        CUDA_CHECK_RETURN(hipMallocManaged(&(m->noise), sizeof(u_real) * m->noise_size));
        for (int i = 0; i < m->noise_size; i++) {
            #ifdef USE_FLOATS
            m->noise[i] = normal_dist(rand_gen);
            #else
            m->noise[i] = (double)normal_dist(rand_gen);
            #endif
        }
        #ifdef NOISE_SEGMENT
        // create shuffled nodes and ts indices for each repeat of the 
        // precalculaed noise 
        if (m->base_conf.verbose) {
            std::cout << "noise will be repeated " << m->noise_repeats << 
                " times (nodes [rows] and timepoints [columns] will be shuffled in each repeat)" << std::endl;
        }
        CUDA_CHECK_RETURN(hipMallocManaged(&(m->shuffled_nodes), sizeof(int) * m->noise_repeats * m->nodes));
        CUDA_CHECK_RETURN(hipMallocManaged(&(m->shuffled_ts), sizeof(int) * m->noise_repeats * m->base_conf.noise_time_steps));
        get_shuffled_nodes_ts(&(m->shuffled_nodes), &(m->shuffled_ts),
            m->nodes, m->base_conf.noise_time_steps, m->noise_repeats, &rand_gen);
        #endif
    } else {
        if (m->base_conf.verbose) {
            std::cout << "Noise already precalculated" << std::endl;
        }
    }

    m->gpu_initialized = true;
}

void BaseModel::free_gpu() {
    if (strcmp(this->get_name(), "Base")==0) {
        // skip freeing memory for BaseModel
        // though free_gpu normally is not called for BaseModel
        // but keeping it here for safety
        return;
    }
    if (!this->gpu_initialized) {
        // if gpu not initialized, skip freeing memory
        return;
    }
    if (this->base_conf.verbose) {
        std::cout << "Freeing GPU memory (" << this->get_name() << ")" << std::endl;
    }
    #ifdef NOISE_SEGMENT
    CUDA_CHECK_RETURN(hipFree(this->shuffled_nodes));
    CUDA_CHECK_RETURN(hipFree(this->shuffled_ts));
    #endif
    CUDA_CHECK_RETURN(hipFree(this->noise));
    for (int sim_idx=0; sim_idx<this->N_SIMS; sim_idx++) {
        #ifdef USE_FLOATS
        CUDA_CHECK_RETURN(hipFree(this->d_fcd_trils[sim_idx]));
        CUDA_CHECK_RETURN(hipFree(this->d_fc_trils[sim_idx]));
        #endif
        CUDA_CHECK_RETURN(hipFree(this->fcd_trils[sim_idx]));
        CUDA_CHECK_RETURN(hipFree(this->windows_ssd_fc[sim_idx]));
        CUDA_CHECK_RETURN(hipFree(this->windows_mean_fc[sim_idx]));
        CUDA_CHECK_RETURN(hipFree(this->windows_fc_trils[sim_idx]));
        CUDA_CHECK_RETURN(hipFree(this->windows_ssd_bold[sim_idx]));
        CUDA_CHECK_RETURN(hipFree(this->windows_mean_bold[sim_idx]));
        CUDA_CHECK_RETURN(hipFree(this->fc_trils[sim_idx]));
        CUDA_CHECK_RETURN(hipFree(this->ssd_bold[sim_idx]));
        CUDA_CHECK_RETURN(hipFree(this->mean_bold[sim_idx]));
        CUDA_CHECK_RETURN(hipFree(this->BOLD[sim_idx]));
    }
    #ifdef USE_FLOATS
    CUDA_CHECK_RETURN(hipFree(this->d_fcd_trils));
    CUDA_CHECK_RETURN(hipFree(this->d_fc_trils));
    #endif
    CUDA_CHECK_RETURN(hipFree(this->fcd_trils));
    CUDA_CHECK_RETURN(hipFree(this->window_pairs_j));
    CUDA_CHECK_RETURN(hipFree(this->window_pairs_i));
    CUDA_CHECK_RETURN(hipFree(this->windows_ssd_fc));
    CUDA_CHECK_RETURN(hipFree(this->windows_mean_fc));
    CUDA_CHECK_RETURN(hipFree(this->windows_fc_trils));
    CUDA_CHECK_RETURN(hipFree(this->windows_ssd_bold));
    CUDA_CHECK_RETURN(hipFree(this->windows_mean_bold));
    CUDA_CHECK_RETURN(hipFree(this->window_ends));
    CUDA_CHECK_RETURN(hipFree(this->window_starts));
    CUDA_CHECK_RETURN(hipFree(this->pairs_j));
    CUDA_CHECK_RETURN(hipFree(this->pairs_i));
    CUDA_CHECK_RETURN(hipFree(this->fc_trils));
    CUDA_CHECK_RETURN(hipFree(this->ssd_bold));
    CUDA_CHECK_RETURN(hipFree(this->mean_bold));
    CUDA_CHECK_RETURN(hipFree(this->BOLD));
    if (this->base_conf.ext_out) {
        for (int var_idx=0; var_idx<this->get_n_state_vars(); var_idx++) {
            for (int sim_idx=0; sim_idx<this->N_SIMS; sim_idx++) {
                CUDA_CHECK_RETURN(hipFree(this->states_out[var_idx][sim_idx]));
            }
            CUDA_CHECK_RETURN(hipFree(this->states_out[var_idx]));
        }
        CUDA_CHECK_RETURN(hipFree(this->states_out));
    }
    if (this->get_n_global_out_bool() > 0) {
        for (int i=0; i<this->get_n_global_out_bool(); i++) {
            CUDA_CHECK_RETURN(hipFree(this->global_out_bool[i]));
        }
        CUDA_CHECK_RETURN(hipFree(this->global_out_bool));
    }
    if (this->get_n_global_out_int() > 0) {
        for (int i=0; i<this->get_n_global_out_int(); i++) {
            CUDA_CHECK_RETURN(hipFree(this->global_out_int[i]));
        }
        CUDA_CHECK_RETURN(hipFree(this->global_out_int));
    }
    if (this->get_n_regional_params() > 0) {
        for (int i=0; i<this->get_n_regional_params(); i++) {
            CUDA_CHECK_RETURN(hipFree(this->d_regional_params[i]));
        }
        CUDA_CHECK_RETURN(hipFree(this->d_regional_params));
    }
    if (this->get_n_global_params() > 0) {
        for (int i=0; i<this->get_n_global_params(); i++) {
            CUDA_CHECK_RETURN(hipFree(this->d_global_params[i]));
        }
        CUDA_CHECK_RETURN(hipFree(this->d_global_params));
    }
    CUDA_CHECK_RETURN(hipFree(this->d_SC_indices));
    for (int SC_idx=0; SC_idx<this->N_SCs; SC_idx++) {
        CUDA_CHECK_RETURN(hipFree(this->d_SC[SC_idx]));
    }
    CUDA_CHECK_RETURN(hipFree(this->d_SC));
}
