#include "hip/hip_runtime.h"
#include "cubnm/includes.cuh"
#include "cubnm/defines.h"
#include "cubnm/fc.cuh"
__global__ void bold_stats(
    u_real **mean_bold, u_real **ssd_bold,
    u_real **BOLD, int N_SIMS, int nodes,
    int bold_len, int corr_len, int n_vols_remove) {
    // TODO: consider combining this with window_bold_stats
    // get simulation index
    int sim_idx = blockIdx.x;
    if (sim_idx >= N_SIMS) return;
    // get node index
    int j = threadIdx.x;
    if (j >= nodes) return;

    // mean
    u_real _mean_bold = 0;
    int vol;
    for (vol=n_vols_remove; vol<bold_len; vol++) {
        _mean_bold += BOLD[sim_idx][vol*nodes+j];
    }
    _mean_bold /= corr_len;
    // ssd
    u_real _ssd_bold = 0;
    for (vol=n_vols_remove; vol<bold_len; vol++) {
        _ssd_bold += POW(BOLD[sim_idx][vol*nodes+j] - _mean_bold, 2);
    }
    // save to memory
    mean_bold[sim_idx][j] = _mean_bold;
    ssd_bold[sim_idx][j] = SQRT(_ssd_bold);
}

__global__ void window_bold_stats(
    u_real **BOLD, int N_SIMS, int nodes,
    int n_windows, int window_size_1, int *window_starts, int *window_ends,
    u_real **windows_mean_bold, u_real **windows_ssd_bold) {
        // get simulation index
        int sim_idx = blockIdx.x;
        if (sim_idx >= N_SIMS) return;
        // get window index
        int w = blockIdx.y;
        if (w >= n_windows) return;
        // get node index
        int j = threadIdx.x;
        if (j >= nodes) return;
        // calculate mean of window
        u_real _mean_bold = 0;
        int vol;
        for (vol=window_starts[w]; vol<=window_ends[w]; vol++) {
            _mean_bold += BOLD[sim_idx][vol*nodes+j];
        }
        _mean_bold /= window_size_1;
        // calculate sd of window
        u_real _ssd_bold = 0;
        for (vol=window_starts[w]; vol<=window_ends[w]; vol++) {
            _ssd_bold += POW(BOLD[sim_idx][vol*nodes+j] - _mean_bold, 2);
        }
        // save to memory
        windows_mean_bold[sim_idx][w*nodes+j] = _mean_bold;
        windows_ssd_bold[sim_idx][w*nodes+j] = SQRT(_ssd_bold);
}

__global__ void fc(u_real **fc_trils, u_real **windows_fc_trils,
    u_real **BOLD, int N_SIMS, int nodes, int n_pairs, int *pairs_i,
    int *pairs_j, int bold_len, int n_vols_remove, 
    int corr_len, u_real **mean_bold, u_real **ssd_bold, 
    int n_windows, int window_size_1, u_real **windows_mean_bold, u_real **windows_ssd_bold,
    int *window_starts, int *window_ends,
    int maxThreadsPerBlock) {
        // get simulation index
        int sim_idx = blockIdx.x;
        if (sim_idx >= N_SIMS) return;
        // get pair index
        int pair_idx = threadIdx.x + (maxThreadsPerBlock * blockIdx.y);
        if (pair_idx >= n_pairs) return;
        int i = pairs_i[pair_idx];
        int j = pairs_j[pair_idx];
        // get window index
        int w = blockIdx.z - 1; // -1 indicates total FC
        if (w >= n_windows) return;
        int vol_start, vol_end;
        u_real _mean_bold_i, _mean_bold_j, _ssd_bold_i, _ssd_bold_j;
        if (w == -1) {
            vol_start = n_vols_remove;
            vol_end = bold_len;
            _mean_bold_i = mean_bold[sim_idx][i];
            _ssd_bold_i = ssd_bold[sim_idx][i];
            _mean_bold_j = mean_bold[sim_idx][j];
            _ssd_bold_j = ssd_bold[sim_idx][j];
        } else {
            vol_start = window_starts[w];
            vol_end = window_ends[w]+1; // +1 because end is non-inclusive
            _mean_bold_i = windows_mean_bold[sim_idx][w*nodes+i];
            _ssd_bold_i = windows_ssd_bold[sim_idx][w*nodes+i];
            _mean_bold_j = windows_mean_bold[sim_idx][w*nodes+j];
            _ssd_bold_j = windows_ssd_bold[sim_idx][w*nodes+j];
        }
        // calculate sigma(x_i * x_j)
        int vol;
        u_real cov = 0;
        for (vol=vol_start; vol<vol_end; vol++) {
            cov += (BOLD[sim_idx][vol*nodes+i] - _mean_bold_i) * (BOLD[sim_idx][vol*nodes+j] - _mean_bold_j);
        }
        // calculate corr(i, j)
        u_real corr = cov / (_ssd_bold_i * _ssd_bold_j);
        if (w == -1) {
            fc_trils[sim_idx][pair_idx] = corr;
        } else {
            windows_fc_trils[sim_idx][w*n_pairs+pair_idx] = corr;
        }
    }

__global__ void window_fc_stats(
    u_real **windows_mean_fc, u_real **windows_ssd_fc,
    u_real **L_windows_mean_fc, u_real **L_windows_ssd_fc,
    u_real **R_windows_mean_fc, u_real **R_windows_ssd_fc,
    u_real **windows_fc_trils, int N_SIMS, int n_windows, int n_pairs,
    bool save_hemis, int n_pairs_hemi) {
        // get simulation index
        int sim_idx = blockIdx.x;
        if (sim_idx >= N_SIMS) return;
        // get window index
        int w = threadIdx.x;
        if (w >= n_windows) return;
        // get hemi
        int hemi = blockIdx.z;
        if (!save_hemis) {
            if (hemi > 0) return;
        } else {
            if (hemi > 2) return;
        }
        // calculate mean fc of window
        u_real _mean_fc = 0;
        int pair_idx_start = 0;
        int pair_idx_end = n_pairs; // non-inclusive
        int pair_idx;
        int _curr_n_pairs = n_pairs;
        // for left and right specify start and end indices
        // that belong to current hemi. Note that this will work
        // regardless of exc_interhemispheric true or false
        if (hemi == 1) { // left
            pair_idx_end = n_pairs_hemi;
            _curr_n_pairs = n_pairs_hemi;
        } else if (hemi == 2) { // right
            pair_idx_start = n_pairs - n_pairs_hemi;
            _curr_n_pairs = n_pairs_hemi;
        }
        for (pair_idx=pair_idx_start; pair_idx<pair_idx_end; pair_idx++) {
            _mean_fc += windows_fc_trils[sim_idx][w*n_pairs+pair_idx];
        }
        _mean_fc /= _curr_n_pairs;
        // calculate ssd fc of window
        u_real _ssd_fc = 0;
        for (pair_idx=pair_idx_start; pair_idx<pair_idx_end; pair_idx++) {
            _ssd_fc += POW(windows_fc_trils[sim_idx][w*n_pairs+pair_idx] - _mean_fc, 2);
        }
        // save to memory
        if (hemi == 0) {
            windows_mean_fc[sim_idx][w] = _mean_fc;
            windows_ssd_fc[sim_idx][w] = SQRT(_ssd_fc);
        } else if (hemi == 1) {
            L_windows_mean_fc[sim_idx][w] = _mean_fc;
            L_windows_ssd_fc[sim_idx][w] = SQRT(_ssd_fc);
        } else if (hemi == 2) {
            R_windows_mean_fc[sim_idx][w] = _mean_fc;
            R_windows_ssd_fc[sim_idx][w] = SQRT(_ssd_fc);
        }
    }

__global__ void fcd(
    u_real **fcd_trils, u_real **L_fcd_trils, u_real **R_fcd_trils,
    u_real **windows_fc_trils,
    u_real **windows_mean_fc, u_real **windows_ssd_fc,
    u_real **L_windows_mean_fc, u_real **L_windows_ssd_fc,
    u_real **R_windows_mean_fc, u_real **R_windows_ssd_fc,
    int N_SIMS, int n_pairs, int n_windows, int n_window_pairs, 
    int *window_pairs_i, int *window_pairs_j, int maxThreadsPerBlock,
    bool save_hemis, int n_pairs_hemi) {
        // get simulation index
        int sim_idx = blockIdx.x;
        if (sim_idx >= N_SIMS) return;
        // get window pair index
        int window_pair_idx = threadIdx.x + (maxThreadsPerBlock * blockIdx.y);
        if (window_pair_idx >= n_window_pairs) return;
        int w_i = window_pairs_i[window_pair_idx];
        int w_j = window_pairs_j[window_pair_idx];
        // get hemi
        int hemi = blockIdx.z;
        if (!save_hemis) {
            if (hemi > 0) return;
        } else {
            if (hemi > 2) return;
        }
        // calculate cov
        int pair_idx;
        u_real cov = 0;
        // pair_idx_start = 0;
        // pair_idx_end = n_pairs; // non-inclusive
        // if (hemi == 1) { // left
        //     pair_idx_end = n_pairs_hemi;
        // } else if (hemi == 2) { // right
        //     pair_idx_start = n_pairs - n_pairs_hemi;
        // }
        if (hemi == 0) {
            for (pair_idx=0; pair_idx<n_pairs; pair_idx++) {
                cov += 
                    (windows_fc_trils[sim_idx][w_i*n_pairs+pair_idx] - windows_mean_fc[sim_idx][w_i]) 
                    * (windows_fc_trils[sim_idx][w_j*n_pairs+pair_idx] - windows_mean_fc[sim_idx][w_j]);
            }
            fcd_trils[sim_idx][window_pair_idx] = cov / (windows_ssd_fc[sim_idx][w_i] * windows_ssd_fc[sim_idx][w_j]);
        } else if (hemi == 1) {
            for (pair_idx=0; pair_idx<n_pairs_hemi; pair_idx++) {
                cov += 
                    (windows_fc_trils[sim_idx][w_i*n_pairs+pair_idx] - L_windows_mean_fc[sim_idx][w_i]) 
                    * (windows_fc_trils[sim_idx][w_j*n_pairs+pair_idx] - L_windows_mean_fc[sim_idx][w_j]);
            }
            L_fcd_trils[sim_idx][window_pair_idx] = cov / (L_windows_ssd_fc[sim_idx][w_i] * L_windows_ssd_fc[sim_idx][w_j]);
        } else if (hemi == 2) {
            for (pair_idx=n_pairs-n_pairs_hemi; pair_idx<n_pairs; pair_idx++) {
                cov += 
                    (windows_fc_trils[sim_idx][w_i*n_pairs+pair_idx] - R_windows_mean_fc[sim_idx][w_i]) 
                    * (windows_fc_trils[sim_idx][w_j*n_pairs+pair_idx] - R_windows_mean_fc[sim_idx][w_j]);
            }
            R_fcd_trils[sim_idx][window_pair_idx] = cov / (R_windows_ssd_fc[sim_idx][w_i] * R_windows_ssd_fc[sim_idx][w_j]);
        }
    }